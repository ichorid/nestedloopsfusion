#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#define BLOCKSIZE 5
typedef unsigned short uint16;

__device__ unsigned int rolMaskA(unsigned int value, unsigned int amount) {
    return (value << amount) | (value >> ((32 - amount) & 31));
}

__device__ unsigned int lanemask_eq() {
    unsigned int mask;
    asm("mov.u32 %0, %%lanemask_eq;" : "=r"(mask));
    return mask;
}

__device__ unsigned int lfsr(uint16 start_state, int cycles = 1) {
    uint16 lfsr = start_state;
    uint16 bit; /* Must be 16bit to allow bit<<15 later in the code */
    for (int i = 0; i < cycles; i++) {
        /* taps: 16 14 13 11; feedback polynomial: x^16
         * + x^14 + x^13 + x^11 + 1 */
        bit = ((lfsr >> 0) ^ (lfsr >> 2) ^ (lfsr >> 3) ^ (lfsr >> 5)) & 1;
        lfsr = (lfsr >> 1) | (bit << 15);
    }
    return lfsr;
}
__global__ void kernel_gpu(
        int *out,
        unsigned int seed,
        int external_loop_iterations,
        unsigned int exit_mask_width,
        const int int_lfsr_cycles,
        const int repeat_times) {
    const unsigned tid = (blockIdx.x << BLOCKSIZE) + threadIdx.x;
    const unsigned int lmask = lanemask_eq();
    int total_cnt = 0;
    const unsigned int tmask = 0xffffffff >> (32 - exit_mask_width);
    for (int k=0; k < repeat_times; k++){
        uint16 rnd_common = lfsr(seed + 3457, 100);
        uint16 o_cnt = 0;
        do {
            int i_cnt = 0;
            o_cnt++;
            rnd_common = lfsr(rnd_common);
            const unsigned int exit_mask = rolMaskA(tmask, rnd_common & 31);

            uint16 irnd = lfsr(rnd_common ^ tid + 321, 3);
            do {
                i_cnt++;
                total_cnt += (irnd & 1);
                irnd = lfsr(irnd, int_lfsr_cycles);
            } while (!((lmask & exit_mask)) && (i_cnt < 10000));
        } while (o_cnt < external_loop_iterations);
    }
    out[tid] = total_cnt;
}

int main(int argc, char *argv[]) {

    int repeat_times = atoi(argv[1]);
    uint16 seed = atoi(argv[2]);
    int external_loop_iterations = atoi(argv[3]);
    unsigned int zt = strtoul(argv[4], NULL, 0);
    int int_lfsr_cycles = atoi(argv[5]);

    int *D_out;

    int num_blocks = 1 << 0;
    int num_threads = num_blocks * (1 << BLOCKSIZE);
    hipEvent_t start, stop;
    checkCudaErrors(hipMalloc((void **)&D_out, num_threads * sizeof(int)));
    hipEventCreate(&start);
    hipEventRecord(start, 0);
    kernel_gpu<<<num_blocks, (1 << BLOCKSIZE)>>>(D_out, seed, external_loop_iterations, zt,
            int_lfsr_cycles, repeat_times);
    hipDeviceSynchronize();
    hipEventCreate(&stop);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    double tm = elapsedTime / 1000;

    int *H_out = (int *)malloc(num_threads * sizeof(int));
    checkCudaErrors(hipMemcpy((void *)H_out, D_out, num_threads * sizeof(int),
                hipMemcpyDeviceToHost));
    long int sum = 0;
    for (int i = 0; i < num_threads; i++) {
        sum += H_out[i];
        printf("\n %i", H_out[i]);
    }
    printf("\n Time Sum Avg Avgt/elem %f %li %li %f", tm, sum, sum / num_threads,
            sum / num_threads / tm);
    printf("\n");
}
